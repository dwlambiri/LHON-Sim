
#include <hip/hip_runtime.h>
﻿
#define diff_live_index 1
#define diff_bound_index 2
#define diff_dead_index 3
#define diff_extra_index 4
//#define rate_dimensions 4

extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_rate_extra, float k_detox_extra,
	unsigned char* pix_out_of_nerve, unsigned char* rate, float* detox, unsigned int rate_dimensions)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xyN = xy * rate_dimensions;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;

		if (!outside)
		{
			rate[xyN] = diff_extra_index;
			rate[xyN + 1] = diff_extra_index;
			rate[xyN + 2] = diff_extra_index;
			rate[xyN + 3] = diff_extra_index;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = diff_extra_index;
				rate[xyN + 5] = diff_extra_index;
			}
			

			detox[xy] = k_detox_extra;

			if (pix_out_of_nerve[xy + im_size]) rate[xyN] = 0;
			if (pix_out_of_nerve[xy - im_size]) rate[xyN + 1] = 0;
			if (pix_out_of_nerve[xy + 1])		rate[xyN + 2] = 0;
			if (pix_out_of_nerve[xy - 1])		rate[xyN + 3] = 0;
		}
		else {
			rate[xyN] = 0;
			rate[xyN + 1] = 0;
			rate[xyN + 2] = 0;
			rate[xyN + 3] = 0;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = 0;
				rate[xyN + 5] = 0;
			}
		}
	}
}

//bool[,] pix_out_of_nerve = new bool[im_size, im_size];
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int dx = x - nerve_cent_pix;
//        int dy = y - nerve_cent_pix;
//        int dis2 = dx * dx + dy * dy;

//        bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
//        pix_out_of_nerve[x, y] = outside;
//        if (!outside)
//        {
//            pix_idx[pix_idx_num++] = x * im_size + y;
//            for (uint k = 0; k < 4; k++)
//                rate[x, y, k] = k_rate_extra;
//            detox[x, y] = k_detox_extra;
//        }
//    }

