
#include <hip/hip_runtime.h>

#define diff_live_index 1
#define diff_bound_index 2
#define diff_dead_index 3
#define diff_extra_index 4

extern "C" __global__  void cuda_update_live(int n_axons, float* tox, unsigned char* rate, float* detox, float* tox_prod, float on_death_tox, float k_detox_extra, float* death_tox_thres,
	unsigned int * axons_cent_pix, unsigned int* axons_inside_pix, unsigned int* axons_inside_pix_idx, unsigned int* axon_surr_rate, unsigned int* axon_surr_rate_idx,
	bool* axon_is_alive, unsigned char* axon_mask, int* num_alive_axons, int* death_itr, int iteration, int offset)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	if (n < n_axons)
	{
		// [DWL]: I made death_tox_threshold to be an array INSTEAD of a constant 
		//			This way we can set the death threshold DIFFERENTLY for each axon
		//			The death is calculated at the head of the axon
		if (axon_is_alive[n] && tox[offset+axons_cent_pix[n]] > death_tox_thres[n])
		{ 	// Kill the axon
			for (int p = axons_inside_pix_idx[n]; p < axons_inside_pix_idx[n + 1]; p++)
			{
				int idx = axons_inside_pix[p];

				detox[idx] = k_detox_extra;
				tox[offset+idx] += on_death_tox;
				tox_prod[idx] = 0;
				axon_mask[idx] = 2; // dead
			}

			for (int p = axon_surr_rate_idx[n]; p < axon_surr_rate_idx[n + 1]; p++)
				rate[axon_surr_rate[p]] = diff_dead_index;
			
			/*
			int idx4 = 4 * idx;
			rate[idx4] = k_rate_dead_axon;
			rate[idx4 + 1] = k_rate_dead_axon;
			rate[idx4 + 2] = k_rate_dead_axon;
			rate[idx4 + 3] = k_rate_dead_axon;
			*/

			axon_is_alive[n] = false;
			death_itr[n] = iteration;
			atomicAdd(&num_alive_axons[0], -1);
		}
	}
}

