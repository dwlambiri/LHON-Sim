
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_tox_sum(int* pix_idx, int pix_idx_num, float* tox, float* tox_sum, int offset, int imsquare, int no3dLayers)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];
		if (no3dLayers) {
			int sum = 0;
			for (int i = 0; i < no3dLayers; i++) {
				sum += tox[((offset + i) % (no3dLayers + 2))*imsquare + xy];
			}
			atomicAdd(tox_sum, sum);
		}
		else {
			atomicAdd(tox_sum, tox[offset*imsquare + xy]);
		}
		
	}
}
