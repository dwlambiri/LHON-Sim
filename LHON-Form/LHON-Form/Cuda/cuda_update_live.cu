
#include <hip/hip_runtime.h>

#define diff_dead_index 4

/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
private readonly byte diff_zero_index = 0;
private readonly byte diff_live_index = 1;
private readonly byte diff_bound_index_a2e = 2;
private readonly byte diff_bound_index_e2a = 3;
private readonly byte diff_dead_index = 4;
private readonly byte diff_extra_index = 5;
private readonly byte diff_one_index = 6;
*/

extern "C" __global__  void cuda_update_live(int n_axons, float* tox, unsigned char* rate, float* detox, float* tox_prod, float on_death_tox, float k_detox_extra, float* death_tox_thres,
	unsigned int * axons_cent_pix, unsigned int* axons_inside_pix, unsigned int* axons_inside_pix_idx, unsigned int* axon_surr_rate, unsigned int* axon_surr_rate_idx,
	bool* axon_is_alive, unsigned char* axon_mask, int* num_alive_axons, int* death_itr, int iteration, int offset, int pixelNeighbourNumbers)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;

	if (n < n_axons)
	{
		// [DWL]: I made death_tox_threshold to be an array INSTEAD of a constant 
		//			This way we can set the death threshold DIFFERENTLY for each axon
		//			The death is calculated at the head of the axon
		if (axon_is_alive[n] && tox[offset+axons_cent_pix[n]] >= death_tox_thres[n])
		{ 	// Kill the axon
			for (int p = axons_inside_pix_idx[n]; p < axons_inside_pix_idx[n + 1]; p++)
			{
				int idx = axons_inside_pix[p];

				detox[idx] = k_detox_extra;
				tox[offset+idx] += on_death_tox;
				tox_prod[idx] = 0;
				axon_mask[idx] = 2; // dead
				
				/*
				int idxN = pixelNeighbourNumbers * idx;
				for (int i = 0; i < pixelNeigbourNumbers; i++) {
					rate[idxN + i] = diff_dead_index;
				}
				*/
				
			}

			for (int p = axon_surr_rate_idx[n]; p < axon_surr_rate_idx[n + 1]; p++)
				rate[axon_surr_rate[p]] = diff_dead_index;
			
			
			axon_is_alive[n] = false;
			death_itr[n] = iteration;
			atomicAdd(&num_alive_axons[0], -1);
		}
	}
}

