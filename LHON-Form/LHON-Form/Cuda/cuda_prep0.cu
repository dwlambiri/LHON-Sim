
#include <hip/hip_runtime.h>
﻿
#define diff_live_index 1
#define diff_bound_index 2
#define diff_dead_index 3
#define diff_extra_index 4

extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_rate_extra, float k_detox_extra,
	unsigned char* pix_out_of_nerve, unsigned char* rate, float* detox)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xy4 = xy * 4;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;

		if (!outside)
		{
			rate[xy4] = diff_extra_index;
			rate[xy4 + 1] = diff_extra_index;
			rate[xy4 + 2] = diff_extra_index;
			rate[xy4 + 3] = diff_extra_index;

			detox[xy] = k_detox_extra;

			if (pix_out_of_nerve[xy + im_size]) rate[xy4] = 0;
			if (pix_out_of_nerve[xy - im_size]) rate[xy4 + 1] = 0;
			if (pix_out_of_nerve[xy + 1])		rate[xy4 + 2] = 0;
			if (pix_out_of_nerve[xy - 1])		rate[xy4 + 3] = 0;
		}
		else {
			rate[xy4] = 0;
			rate[xy4 + 1] = 0;
			rate[xy4 + 2] = 0;
			rate[xy4 + 3] = 0;
		}
	}
}

//bool[,] pix_out_of_nerve = new bool[im_size, im_size];
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int dx = x - nerve_cent_pix;
//        int dy = y - nerve_cent_pix;
//        int dis2 = dx * dx + dy * dy;

//        bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
//        pix_out_of_nerve[x, y] = outside;
//        if (!outside)
//        {
//            pix_idx[pix_idx_num++] = x * im_size + y;
//            for (uint k = 0; k < 4; k++)
//                rate[x, y, k] = k_rate_extra;
//            detox[x, y] = k_detox_extra;
//        }
//    }

