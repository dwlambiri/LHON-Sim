
#include <hip/hip_runtime.h>
﻿
#define diff_extra_index 5
#define diff_extra_z_index 7

/*
* [DWL] Need to keep the constants in this file in line with the values in preprocess.cs!!
		private readonly byte diff_zero_index = 0;
		private readonly byte diff_live_index = 1;
		private readonly byte diff_bound_index_a2e = 2;
		private readonly byte diff_bound_index_e2a = 3;
		private readonly byte diff_dead_index = 4;
		private readonly byte diff_extra_index = 5;
		private readonly byte diff_live_z_index = 6;
		private readonly byte diff_extra_z_index = 7;
		private readonly byte diff_one_index = 8;
		private readonly byte diff_values_size = 9;
*/

extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_detox_extra,
	unsigned char* pix_out_of_nerve, unsigned char* rate, float* detox, unsigned int rate_dimensions)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xyN = xy * rate_dimensions;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;

		if (!outside)
		{
			rate[xyN] = diff_extra_index;
			rate[xyN + 1] = diff_extra_index;
			rate[xyN + 2] = diff_extra_index;
			rate[xyN + 3] = diff_extra_index;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = diff_extra_z_index;
				rate[xyN + 5] = diff_extra_z_index;
			}
			

			detox[xy] = k_detox_extra;

			if (pix_out_of_nerve[xy + im_size]) rate[xyN] = 0;
			if (pix_out_of_nerve[xy - im_size]) rate[xyN + 1] = 0;
			if (pix_out_of_nerve[xy + 1])		rate[xyN + 2] = 0;
			if (pix_out_of_nerve[xy - 1])		rate[xyN + 3] = 0;
		}
		else {
			rate[xyN] = 0;
			rate[xyN + 1] = 0;
			rate[xyN + 2] = 0;
			rate[xyN + 3] = 0;
			if (rate_dimensions > 4) {
				rate[xyN + 4] = 0;
				rate[xyN + 5] = 0;
			}
		}
	}
}

//bool[,] pix_out_of_nerve = new bool[im_size, im_size];
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int dx = x - nerve_cent_pix;
//        int dy = y - nerve_cent_pix;
//        int dis2 = dx * dx + dy * dy;

//        bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
//        pix_out_of_nerve[x, y] = outside;
//        if (!outside)
//        {
//            pix_idx[pix_idx_num++] = x * im_size + y;
//            for (uint k = 0; k < 4; k++)
//                rate[x, y, k] = k_rate_extra;
//            detox[x, y] = k_detox_extra;
//        }
//    }

