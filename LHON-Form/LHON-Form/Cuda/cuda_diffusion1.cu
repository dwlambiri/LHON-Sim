
#include <hip/hip_runtime.h>
﻿
#define diff_live_index 1
#define diff_bound_index 2
#define diff_dead_index 3
#define diff_extra_index 4

extern "C" __global__  void cuda_diffusion1(int* pix_idx, int pix_idx_num, unsigned short im_size,
	int tox_switch, float* tox, float* detox, float* tox_prod, unsigned char* rate, float* rate_values, int rate_dimensions)
{
	//float rate_values[] = { 0,0,0,0,0,0 };
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		int xy0 = xy + im_size;
		int xy1 = xy - im_size;
		int xy2 = xy + 1;
		int xy3 = xy - 1;
		int xyN = xy * rate_dimensions;

		float *tox_new, *tox_old;

		if (tox_switch > 0) {
			tox_old = &tox[im_size*im_size];
			tox_new = &tox[0];
		}
		else {
			tox_new = &tox[im_size*im_size];
			tox_old = &tox[0];
		}

		float t = tox_old[xy];

		tox_new[xy] = t +
			(tox_old[xy0] - t) * rate_values[rate[xyN]] +
			(tox_old[xy1] - t) * rate_values[rate[xyN + 1]] +
			(tox_old[xy2] - t) * rate_values[rate[xyN + 2]] +
			(tox_old[xy3] - t) * rate_values[rate[xyN + 3]] +
			tox_prod[xy];

		tox_new[xy] *= detox[xy];
	}
}
