
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_update_image(unsigned short im_size, unsigned short bmp_im_size, float bmp_image_compression_ratio,
	unsigned char* bmp, float* tox, unsigned char* axon_mask, unsigned char* init_insult_mask, float tox_max, bool* show_opts, int showdir, int lineToDisplay, int imsq, int head, int no3d)
{
	int x_bmp = blockIdx.x * blockDim.x + threadIdx.x;
	int y_bmp = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_bmp < bmp_im_size && y_bmp > 0) {

		int xy_bmp = x_bmp * bmp_im_size + y_bmp;
		int xy4_bmp = xy_bmp * 4;
		

		unsigned char red = 0, green = 0, blue = 0;
		
		switch (showdir) {
		case 1: { 
			// XZ (vertical slider)
			//green = blue = 0;
			if ( (x_bmp >= lineToDisplay) && (x_bmp < lineToDisplay + no3d)  && show_opts[2]) {
				int layer = (head + x_bmp - lineToDisplay) % (no3d + 2);
				int xpos = (int)((float)(bmp_im_size - lineToDisplay) * bmp_image_compression_ratio);
				int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
				int xy = ypos * im_size + xpos;
				float tmp = tox[xy+ imsq*layer] / tox_max;
				if (tmp > 1) tmp = 1;
				red = (unsigned char)(tmp * 255); // 0 - 255
				//red = 255;
			}
			break;
		}
		case 2: {
			// YZ (horizontal slider)
			//green = blue = 0;
			if ((y_bmp >= lineToDisplay) && (y_bmp < lineToDisplay + no3d) && show_opts[2]) {
				int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
				int ypos = (int)((float)lineToDisplay * bmp_image_compression_ratio);
				int xy = ypos * im_size + xpos;
				int layer = (head + y_bmp - lineToDisplay) % (no3d + 2);
				float tmp = tox[xy+ imsq * layer] / tox_max;
				if (tmp > 1) tmp = 1;
				red = (unsigned char)(tmp * 255); // 0 - 255
				//red = 255;
			}
			break;
		}
		default: {
			int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
			int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
			int xy = ypos * im_size + xpos;
			float tmp = tox[imsq*lineToDisplay + xy] / tox_max;
			if (tmp > 1) tmp = 1;
			unsigned char normalized_toxin = (unsigned char)(tmp * 255); // 0 - 255
			if (init_insult_mask[xy_bmp]) { blue = green = 127; /*red = 0;*/ }
			else
			{
				if (show_opts[0] && show_opts[1]) {
					if (axon_mask[xy] == 1) { green = 100; } // live
					if (axon_mask[xy] == 2) { blue = 255; /*green = 0;*/ } // dead
					// else: the pixel doesn't belongs to any axon
				}
				else if (show_opts[0]) {
					if (axon_mask[xy] == 1) { green = 100; } // live
					//if (axon_mask[xy] == 2) { green = 0; } // dead
					blue = 0;
				}
				else if (show_opts[1]) {
					//if (axon_mask[xy] == 1) { blue = 0; } // live
					if (axon_mask[xy] == 2) { blue = 255; } // dead
					green = 0;
				}
				else {
					//blue = green = 0;
				}

				if (show_opts[2]) {
					red = normalized_toxin;
					// green = 255 - normalized_toxin;
				}
				//else { red = 0; }
			}
		}
		}
		

		bmp[xy4_bmp] = blue;
		bmp[xy4_bmp + 1] = green;
		bmp[xy4_bmp + 2] = red;
	}
}

/*

// Jet colormap: https://www.mathworks.com/help/matlab/ref/jet.html

if (normalized_toxin < 64) { r = 0; g = 4 * v; b = 255; }
else if (normalized_toxin < 128) { r = 0; b = 255 + 4 * (64 - v); g = 255; }
else if (normalized_toxin < 192) { r = 4 * (v - 128); b = 0; g = 255; }
else { g = 255 + 4 * (192 - normalized_toxin); b = 0; r = 255; }

*/
