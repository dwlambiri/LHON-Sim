
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_update_image(unsigned short im_size, unsigned short bmp_im_size, float bmp_image_compression_ratio, float bmp_z_compression_ratio,
	unsigned char* bmp, float* bmp_tox, float* tox, unsigned char* axon_mask, unsigned char* init_insult_mask, float tox_max, bool* show_opts, int showdir, int lineToDisplay, int imsq, int head, int no3d, int showRGBSox, int displayAtTop)
{
	int x_bmp = blockIdx.x * blockDim.x + threadIdx.x;
	int y_bmp = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_bmp < bmp_im_size && y_bmp > 0) {

		int xy_bmp = x_bmp * bmp_im_size + y_bmp;
		int xy4_bmp = xy_bmp * 4;
		

		unsigned char red = 0, green = 0, blue = 0;
		float tox_pix_value = 0;
		
		switch (showdir) {
		case 1: { 
			// XZ (vertical slider)
			//green = blue = 0;

			if (displayAtTop) {
				if (show_opts[2]) {
					float xf = x_bmp * bmp_z_compression_ratio;
					int layer = (head+(int)(xf)) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - lineToDisplay) * bmp_image_compression_ratio);
					int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					red = (unsigned char)(tmp * 255); // 0 - 255
					if (showRGBSox) {
						float gt = tmp * 255 - red;
						if (gt > 0) {
							green = (unsigned char)(gt * 255);
							gt = gt * 255 - green;
							if (gt > 0) {
								blue = (unsigned char)(gt * 255);
							}
						}
					}
				}
			}
			else {
				if ((x_bmp >= lineToDisplay) && (x_bmp < lineToDisplay + no3d) && show_opts[2]) {
					int layer = (head + x_bmp - lineToDisplay) % (no3d + 2);
					int xpos = (int)((float)(bmp_im_size - lineToDisplay) * bmp_image_compression_ratio);
					int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					red = (unsigned char)(tmp * 255); // 0 - 255
					if (showRGBSox) {
						float gt = tmp * 255 - red;
						if (gt > 0) {
							green = (unsigned char)(gt * 255);
							gt = gt * 255 - green;
							if (gt > 0) {
								blue = (unsigned char)(gt * 255);
							}
						}
					}
					//red = 255;
				}
			}
			break;
		}
		case 2: {
			// YZ (horizontal slider)
			//green = blue = 0;
			if (displayAtTop) {
				if (show_opts[2]) {
					int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
					int ypos = (int)((float)lineToDisplay * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					float yf = y_bmp * bmp_z_compression_ratio;
					int layer = (head + (int)(yf)) % (no3d + 2);
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					red = (unsigned char)(tmp * 255); // 0 - 255
					//red = 255;
					red = (unsigned char)(tmp * 255); // 0 - 255
					if (showRGBSox) {
						float gt = tmp * 255 - red;
						if (gt > 0) {
							green = (unsigned char)(gt * 255);
							gt = gt * 255 - green;
							if (gt > 0) {
								blue = (unsigned char)(gt * 255);
							}
						}
					}
				}
			}
			else {
				if ((y_bmp >= lineToDisplay) && (y_bmp < lineToDisplay + no3d) && show_opts[2]) {
					int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
					int ypos = (int)((float)lineToDisplay * bmp_image_compression_ratio);
					int xy = ypos * im_size + xpos;
					//int xy = xpos * im_size + ypos;
					int layer = (head + y_bmp - lineToDisplay) % (no3d + 2);
					tox_pix_value = tox[xy + imsq * layer];
					float tmp = tox_pix_value / tox_max;
					if (tmp > 1) tmp = 1;
					red = (unsigned char)(tmp * 255); // 0 - 255
					//red = 255;
					red = (unsigned char)(tmp * 255); // 0 - 255
					if (showRGBSox) {
						float gt = tmp * 255 - red;
						if (gt > 0) {
							green = (unsigned char)(gt * 255);
							gt = gt * 255 - green;
							if (gt > 0) {
								blue = (unsigned char)(gt * 255);
							}
						}
					}
				}
			}
			break;
		}
		default: {
			int xpos = (int)((float)(bmp_im_size - x_bmp) * bmp_image_compression_ratio);
			int ypos = (int)((float)y_bmp * bmp_image_compression_ratio);
			int xy = ypos * im_size + xpos;
			//int xy = xpos * im_size + ypos;
			tox_pix_value = tox[imsq * lineToDisplay + xy];
			float tmp = tox_pix_value / tox_max;
			if (tmp > 1) tmp = 1;
			unsigned char normalized_toxin = (unsigned char)(tmp * 255); // 0 - 255
			if (init_insult_mask[xy_bmp]) { blue = green = 127; /*red = 0;*/ }
			else
			{
				if (show_opts[0] && show_opts[1]) {
					if (axon_mask[xy] == 1) { green = 100; } // live
					if (axon_mask[xy] == 2) { blue = 255; /*green = 0;*/ } // dead
					// else: the pixel doesn't belongs to any axon
				}
				else if (show_opts[0]) {
					if (axon_mask[xy] == 1) { green = 100; } // live
					//if (axon_mask[xy] == 2) { green = 0; } // dead
					blue = 0;
				}
				else if (show_opts[1]) {
					//if (axon_mask[xy] == 1) { blue = 0; } // live
					if (axon_mask[xy] == 2) { blue = 255; } // dead
					green = 0;
				}
				else {
					//blue = green = 0;
				}

				if (show_opts[2]) {
					if (show_opts[0] == 0 && show_opts[1] == 0) {
						red = normalized_toxin;
						if (showRGBSox) {
							float gt = tmp * 255 - red;
							if (gt > 0) {
								green = (unsigned char)(gt * 255);
								gt = gt * 255 - green;
								if (gt > 0) {
									blue = (unsigned char)(gt * 255);
								}
							}
						}
					}
					else {
						red = normalized_toxin;
					}
					// green = 255 - normalized_toxin;
				}
				//else { red = 0; }
			}
		}
		}
		

		bmp[xy4_bmp] = blue;
		bmp[xy4_bmp + 1] = green;
		bmp[xy4_bmp + 2] = red;

		bmp_tox[xy_bmp] = tox_pix_value;
	}
}

/*

// Jet colormap: https://www.mathworks.com/help/matlab/ref/jet.html

if (normalized_toxin < 64) { r = 0; g = 4 * v; b = 255; }
else if (normalized_toxin < 128) { r = 0; b = 255 + 4 * (64 - v); g = 255; }
else if (normalized_toxin < 192) { r = 4 * (v - 128); b = 0; g = 255; }
else { g = 255 + 4 * (192 - normalized_toxin); b = 0; r = 255; }

*/
