
#include <hip/hip_runtime.h>
﻿
#define rateUpLayerIndex  4
#define rateDownLayerIndex  5

extern "C" __global__  void cuda_diffusion2(int* pix_idx, int pix_idx_num, unsigned short im_size,
	float* tox, float* detox, float* tox_prod, float* randTable, unsigned char* rate, float* rate_values, int rate_dimensions, 
	int dstl, int tl, int ml, int bl, int top, int bottom, int injury, int index)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
	if (idx < pix_idx_num)
	{
		int xy = pix_idx[idx];

		int sq = im_size * im_size;

		int xy0 = xy + im_size;
		int xy1 = xy - im_size;
		int xy2 = xy + 1;
		int xy3 = xy - 1;
		
		int xyN = xy * rate_dimensions;

		float* tox_new = &tox[dstl * sq];
	    float* tox_old = &tox[ml * sq];
		float* tox_up = &tox[tl * sq];
		float* tox_down = &tox[bl * sq];

		float t = tox_old[xy];

		tox_new[xy] = t +
			(tox_old[xy0] - t) * rate_values[rate[xyN]] +
			(tox_old[xy1] - t) * rate_values[rate[xyN + 1]] +
			(tox_old[xy2] - t) * rate_values[rate[xyN + 2]] +
			(tox_old[xy3] - t) * rate_values[rate[xyN + 3]];

		if(top == false)
			tox_new[xy] += (tox_up[xy] - t) * rate_values[rate[xyN + rateUpLayerIndex]];

		if (bottom == false)
			tox_new[xy] += (tox_down[xy] - t) * rate_values[rate[xyN + rateDownLayerIndex]];

		if(injury)
			tox_new[xy] += tox_prod[xy] * randTable[index];

		tox_new[xy] *= detox[xy];
	}
}
