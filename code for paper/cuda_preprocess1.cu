
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_rate_extra, float k_detox_extra,
	unsigned char* pix_out_of_nerve, float* rate, float* detox)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xy4 = xy * 4;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;

		if (!outside)
		{
			rate[xy4] = k_rate_extra;
			rate[xy4 + 1] = k_rate_extra;
			rate[xy4 + 2] = k_rate_extra;
			rate[xy4 + 3] = k_rate_extra;

			detox[xy] = k_detox_extra;
		}
	}
}
