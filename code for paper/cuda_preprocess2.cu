
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_prep1(unsigned short im_size, unsigned char* pix_out_of_nerve, float* rate)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size)
	{
		int xy = x * im_size + y;
		int xy4 = xy * 4;

		if (pix_out_of_nerve[xy]) {
			rate[xy4] = 0;
			rate[xy4 + 1] = 0;
			rate[xy4 + 2] = 0;
			rate[xy4 + 3] = 0;
		}
		else {
			if (pix_out_of_nerve[xy + im_size]) rate[xy4] = 0;
			if (pix_out_of_nerve[xy - im_size]) rate[xy4 + 1] = 0;
			if (pix_out_of_nerve[xy + 1])		rate[xy4 + 2] = 0;
			if (pix_out_of_nerve[xy - 1])		rate[xy4 + 3] = 0;
		}
	}
}
